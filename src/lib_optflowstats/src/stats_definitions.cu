#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "stats_definitions.hpp"
#include <cmath>
#include <chrono>

#define WARP_SIZE 32

// TODO: check possibiities for kernels optimisation
// TODO: check if it is possible to avoid bank conflicts
namespace ofs_detail
{

namespace
{ 
  using cv::cuda::PtrStepSz;
  // using cv::cuda::GpuMat;

__host__ __device__  int nearestGreaterPow2(int n)
{
  --n;
  n |= n >> 1;
  n |= n >> 2;
  n |= n >> 4;
  n |= n >> 8;
  n |= n >> 16;
  return ++n;
} 

__device__ float calc_angle(const float x, const float y)
{
    float angle = atan2f(y, x);
    if (angle < 0)
        angle += 2*CV_PI;
    return angle;
  }

  
__global__ void k_calc_and_part_reduce_HoA(const PtrStepSz<float2> flow, unsigned* hist, const unsigned binsNum)
{
  const int i = blockIdx.y * blockDim.y + threadIdx.y;
  const int j = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ unsigned hist_shared[];

  int numOfcells = binsNum * blockDim.y;
  int numOfThreads = blockDim.y * blockDim.x;
  int numOfIters = numOfcells / numOfThreads;
  int thID = threadIdx.y * blockDim.x + threadIdx.x;

  auto boundaryCheck = (i < flow.rows && j < flow.cols);
// get an address for a given row in thread block
  auto getShMemIndex_y = [](const unsigned index)
  {
      return threadIdx.y + index * blockDim.y;
  };

  for (int i = 0 ; i <= numOfIters; i++) 
  {
    int linearID = i * numOfThreads + thID;
    if (linearID < numOfcells)
    {
      hist_shared[linearID] = 0;
    }
  }
  __syncthreads();

  if (boundaryCheck)
  {
    const float2 fxy = flow(i, j);
    auto indexInHistogram = static_cast<unsigned>(calc_angle(fxy.x, fxy.y)/(2*CV_PI)*binsNum);
    atomicAdd(&hist_shared[getShMemIndex_y(indexInHistogram)], 1);
  }
  __syncthreads();

// Reduce hist_shared 
  const unsigned xstride = binsNum / blockDim.x;
  for (unsigned stride = nearestGreaterPow2(blockDim.y)/2; stride > 0; stride >>= 1)
  {
    if ( (threadIdx.y < stride) && ((threadIdx.y + stride) < blockDim.y) && boundaryCheck)
    {
      for(unsigned it = 0; it <= xstride; it++)
      {
        auto bin = it * blockDim.x + threadIdx.x;
        if(bin < binsNum)
        {
          atomicAdd(&hist_shared[getShMemIndex_y(bin)], hist_shared[threadIdx.y + stride + bin * blockDim.y]);
        }  
      }
    }
    __syncthreads();
  }

// save to global memory (hist)
  if (boundaryCheck)
  {
    for (int i = 0 ; i <= binsNum / numOfThreads; i++) 
    {
      int index = i * numOfThreads + thID;
      if (index < binsNum)
      {
        hist[(blockIdx.x + blockIdx.y * gridDim.x) * binsNum + index] = hist_shared[index * blockDim.y];
      }
    }
  }
}


__global__ void reduce_hist(unsigned* hist, const unsigned binsNum)
{
// TODO: use shared memory and warp shuffle
//  reduce columns
  for (unsigned stride = nearestGreaterPow2(blockDim.x)/2; stride > 0; stride >>= 1)
  {
    if (threadIdx.x < stride && (threadIdx.x + stride) < blockDim.x)
    {
      for (unsigned id = 0; id < binsNum; id++)
      {
        hist[(threadIdx.x + threadIdx.y * blockDim.x) * binsNum + id] += hist[((threadIdx.x + stride) + threadIdx.y * blockDim.x) * binsNum + id];
      }
    }
    __syncthreads();
  }

// reduce rows

  for(unsigned stride = nearestGreaterPow2(blockDim.y)/2; stride > 0; stride >>= 1)
  {
    if(threadIdx.x == 0 && threadIdx.y < stride && threadIdx.y + stride < blockDim.y)  
    {
      for(unsigned id = 0; id < binsNum; id++)
      {
        hist[(threadIdx.y * blockDim.x) * binsNum + id] += hist[((stride + threadIdx.y) * blockDim.x) * binsNum + id];
      }
    }
    __syncthreads();
  }
}

__device__ __inline_hint__ float calc_divergence(auto & flow)
{
  return (flow(threadIdx.y, threadIdx.x+1).x - flow(threadIdx.y, threadIdx.x).x) 
        + (flow(threadIdx.y+1, threadIdx.x).y - flow(threadIdx.y, threadIdx.x).y);
}

// blockDim.x is supposed to be 32
__global__ void k_abs_div_calc_reduce_part(const PtrStepSz<float2> flow, float * divMat)
{
  extern __shared__ float2 shMem[];

  auto shFlow = [] (int i, int j) -> float2& { return shMem[i * (blockDim.x + 1) + j]; };
  auto gFlow = [&flow] (int i, int j) ->const float2& { return flow(i + blockIdx.y * blockDim.y, j + blockIdx.x * blockDim.x); };           
  auto shDiv = [] (int i) -> float& { return ((float*)&shMem[(blockDim.y-1) * blockDim.x + blockDim.x])[i]; };

  // load data to shared memory
  auto y = blockIdx.y * blockDim.y + threadIdx.y;
  auto x = blockIdx.x * blockDim.x + threadIdx.x;

  if (y < flow.rows && x < flow.cols)
  {
    shFlow(threadIdx.y, threadIdx.x) = gFlow(threadIdx.y, threadIdx.x);
  }

  // this is supposed to be the last warp in a block, so threads should not be highly divergent accross warps
  if( threadIdx.y == blockDim.y - 1) 
  {
    if ((blockIdx.x * blockDim.x + blockDim.x) < flow.cols 
      && (blockIdx.y * blockDim.y + threadIdx.x) < flow.rows
      && threadIdx.x < blockDim.y)
      shFlow(threadIdx.x, blockDim.x) = gFlow(threadIdx.x, blockDim.x);
    if((blockIdx.y * blockDim.y + threadIdx.y) < (flow.rows - 1))
      shFlow(threadIdx.y+1, threadIdx.x) = gFlow(threadIdx.y+1, threadIdx.x);
  }
  __syncthreads();


  if(y < (flow.rows-1) && x < (flow.cols-1))
  {
    // calculate divergence
    auto div = abs(calc_divergence(shFlow));
    // reduce warp (horizontally)
    for (unsigned stride = WARP_SIZE/2; stride > 0; stride >>= 1)
    {
        div += __shfl_xor_sync(__activemask(), div, stride); // default width is warpSize
    }

    // save to shared memory
    if(threadIdx.x == 0)
    {
      shDiv(threadIdx.y) = div;
    }
  }
  __syncthreads();

  // reduce vertically (but using only one warp)
  for (unsigned stride = blockDim.y/2; stride > 0; stride >>= 1)
  {
    if (threadIdx.y == 0 && (blockDim.y * blockIdx.y + threadIdx.x + stride) < (flow.rows-1) && threadIdx.x < stride && (threadIdx.x + stride) < blockDim.y)
    {
      shDiv(threadIdx.x) += shDiv(threadIdx.x + stride);
    }
    __syncthreads();
  }
  
  // // save to global memory
  if(threadIdx.x == 0 && threadIdx.y == 0 && y < (flow.rows-1) && x < (flow.cols-1))
  {
    divMat[blockIdx.y * gridDim.x + blockIdx.x] = shDiv(0);
  }

}

// template <typename T>
__global__ void k_abs_divergence_reduce(float * divMat)//, const unsigned d_mem[][], const int2 size)
{
  extern __shared__ float shMem_r[];

  auto shDiv = [] (int i, int j) -> float& { return shMem_r[i * blockDim.x + j]; };
  auto gDiv = [divMat] (int i, int j) -> float& { return divMat[i * blockDim.x + j]; };
  shDiv(threadIdx.y, threadIdx.x) = gDiv(threadIdx.y, threadIdx.x);

  __syncthreads();

// reduce horizontally
  for (unsigned stride = nearestGreaterPow2(blockDim.x)/2; stride > 0; stride >>= 1)
  {
    if (threadIdx.x < stride && (threadIdx.x + stride) < blockDim.x)
    {
      shDiv(threadIdx.y, threadIdx.x) += shDiv(threadIdx.y, threadIdx.x + stride);
    }
    __syncthreads();
  }

// reduce vertically
  for (unsigned stride = nearestGreaterPow2(blockDim.y)/2; stride > 0; stride >>= 1)
  {
    if (threadIdx.y == 0 && threadIdx.x < stride && threadIdx.x + stride < blockDim.y)
    {
      shDiv(threadIdx.x, threadIdx.y) += shDiv(threadIdx.x + stride, threadIdx.y);
    }
    __syncthreads();
  }

  if(threadIdx.x == 0 && threadIdx.y == 0)
  {
    divMat[0] = shDiv(0, 0);
  }
}

} // namespace

HoA calc_hist_of_angles(const Mat & flow,const unsigned binsNum)
{
  assert( 4 == sizeof(unsigned) );

  // Define block and grid dimensions
  dim3 blockDim(WARP_SIZE, 512/WARP_SIZE);
  dim3 gridDim(std::ceil((float)flow.cols/ blockDim.x), std::ceil((float)flow.rows/ blockDim.y));

  // Allocate device memory for hist (space needed for reduction also)
  auto numBlocks = gridDim.x * gridDim.y;
  unsigned * d_hist = nullptr;

  checkCudaErrors(hipMalloc((void**)&d_hist, numBlocks * binsNum * sizeof(unsigned)));
  checkCudaErrors(hipMemset(d_hist, 0, numBlocks *binsNum * sizeof(unsigned)));

  auto singleHistSize = binsNum * sizeof(unsigned);

  // Launch kernel v2
  auto sharedMemSize = blockDim.y * singleHistSize;
  k_calc_and_part_reduce_HoA<<<gridDim, blockDim, sharedMemSize>>>(flow, d_hist, binsNum);
  checkCudaErrors(hipDeviceSynchronize());

  dim3 blockDim_reduce = gridDim;
  reduce_hist<<<1, blockDim_reduce>>>(d_hist, binsNum);
  checkCudaErrors(hipGetLastError());
  // Copy result back to host
  HoA hist(binsNum);
  checkCudaErrors(hipMemcpy(&hist[0], d_hist, binsNum * sizeof(unsigned), hipMemcpyDeviceToHost));

  // Cleanup
  checkCudaErrors(hipFree(d_hist));
  return hist;
}

MADiv calc_mean_abs_divergence(const Mat & flow)
{
  auto start = std::chrono::high_resolution_clock::now();

  dim3 childBlockDim(WARP_SIZE, WARP_SIZE);
  dim3 blockDim;
  blockDim.x = std::ceil((float)flow.cols/childBlockDim.x); 
  blockDim.y = std::ceil((float)flow.rows/childBlockDim.y);

  float * d_divMat = nullptr;
  checkCudaErrors(hipMalloc((void**)&d_divMat, blockDim.x * blockDim.y * sizeof(float)));

  auto shMem4flow = (childBlockDim.x + 1) * childBlockDim.y + childBlockDim.x;
  auto shMem4div = childBlockDim.y;
  auto shMemSize = shMem4flow * sizeof(float2) + shMem4div * sizeof(float);

  k_abs_div_calc_reduce_part<<<blockDim, childBlockDim, shMemSize>>>(flow, d_divMat);
  checkCudaErrors(hipDeviceSynchronize());

  // TODO: size of shared memory can be lowered at least by a half
  //  using warp shuffle and performing first step of reduction and reading global memoru in the same thread 
  k_abs_divergence_reduce<<<1, blockDim, blockDim.x * blockDim.y * sizeof(float)>>>(d_divMat);
  checkCudaErrors(hipGetLastError());

  float div = 0.0;
  checkCudaErrors(hipMemcpy(&div, d_divMat, sizeof(float), hipMemcpyDeviceToHost));

  float mean = div / ((flow.rows - 1) * (flow.cols - 1));

  return MADiv(mean);
}

} // namespace ofs_detail
